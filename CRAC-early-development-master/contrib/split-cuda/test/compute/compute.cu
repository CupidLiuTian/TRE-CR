#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


extern "C" void cudaFun(int devid, int is, int nx ,int nz)
{
    float *vp_device , *vp_host;
    hipError_t cudaStatus;
    int dev_cnt;
    hipDeviceProp_t prop;
    int dev;
    float mstimer;

    printf("[XB2] cuda start. \n");
    cudaStatus = hipGetDeviceCount(&dev_cnt);
    if(cudaStatus != hipSuccess)
    {
        printf("hipGetDeviceCount failed! \n");
        return ;
    }
    printf("[XB2] The number of gpu is %d. \n", dev_cnt);
    sleep(3);
	printf("Dev: %d, GPU: %d\n", devid, devid % dev_cnt);
	hipSetDevice(devid % dev_cnt);
	hipGetDevice(&dev);
	hipGetDeviceProperties(&prop, dev);
	printf("Name:                     %s\n", prop.name);
	hipMalloc(&vp_device, nx*nz*sizeof(float));  
	hipMemset(vp_device, 0, nx*nz*sizeof(float));

	vp_host=(float*)malloc(nx*nz*sizeof(float));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

  //  sleep(3);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mstimer, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(vp_device);
	free(vp_host);

    printf("[XB2] Finished... \n");
}
